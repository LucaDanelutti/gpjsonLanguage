
#include <hip/hip_runtime.h>
__global__ void sum1(char *charArr, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elems_per_thread = (n+stride-1) / stride;

    long start = index * elems_per_thread;
    long end = start + elems_per_thread;

    char sum = 0;
    for (long i = start; i < end && i < n; i++) {
        sum += charArr[i];
        charArr[i] = sum;
    }
}