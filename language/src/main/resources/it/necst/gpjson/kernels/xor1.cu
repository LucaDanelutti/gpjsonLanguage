
#include <hip/hip_runtime.h>
__global__ void xor1(char *arr, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elems_per_thread = (n+stride-1) / stride;

    long start = index * elems_per_thread;
    long end = start + elems_per_thread;

    char prev = 0;
    for (long i = start; i < end && i < n; i++) {
        prev ^= arr[i];
        arr[i] = prev;
    }
}