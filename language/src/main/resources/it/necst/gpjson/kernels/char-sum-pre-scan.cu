
#include <hip/hip_runtime.h>
__global__ void f(char *charArr, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elemsPerThread = (n+stride-1) / stride;

    long start = index * elemsPerThread;
    long end = start + elemsPerThread;

    char sum = 0;
    for (long i = start; i < end && i < n; i++) {
        sum += charArr[i];
        charArr[i] = sum;
    }
}