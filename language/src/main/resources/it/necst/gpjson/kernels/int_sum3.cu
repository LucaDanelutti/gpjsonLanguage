
#include <hip/hip_runtime.h>
__global__ void sum3(int *arr, int n, int *base, int offset, int *newArr) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elems_per_thread = (n+stride-1) / stride;

    long start = index * elems_per_thread;
    long end = start + elems_per_thread;

    for (long i = start; i < end && i < n; i++) {
        newArr[i+offset] = arr[i] + base[index];
    }
}