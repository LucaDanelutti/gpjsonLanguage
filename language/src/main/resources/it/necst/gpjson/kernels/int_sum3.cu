
#include <hip/hip_runtime.h>
__global__ void sum3(int *intArr, int n, int *base, int offset, int *intNewArr) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elems_per_thread = (n+stride-1) / stride;

    long start = index * elems_per_thread;
    long end = start + elems_per_thread;

    for (long i = start; i < end && i < n; i++) {
        intNewArr[i+offset] = intArr[i] + base[index];
    }
}