
#include <hip/hip_runtime.h>
__global__ void f(char *charArr, int n, char *base) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elemsPerThread = (n+stride-1) / stride;

    long start = index * elemsPerThread;
    long end = start + elemsPerThread;

    for (long i = start; i < end && i < n; i++) {
        charArr[i] ^= base[index];
    }
}