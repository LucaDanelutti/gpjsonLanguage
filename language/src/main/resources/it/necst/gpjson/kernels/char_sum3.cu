
#include <hip/hip_runtime.h>
__global__ void sum3(char *arr, int n, char *base, int offset, char *newArr) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elems_per_thread = (n+stride-1) / stride;

    long start = index * elems_per_thread;
    long end = start + elems_per_thread;

    for (long i = start; i < end && i < n; i++) {
        newArr[i+offset] = arr[i] + base[index];
    }
}