
#include <hip/hip_runtime.h>
__global__ void sum3(char *charArr, int n, char *base, int offset, char *charNewArr) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    long elems_per_thread = (n+stride-1) / stride;

    long start = index * elems_per_thread;
    long end = start + elems_per_thread;

    for (long i = start; i < end && i < n; i++) {
        charNewArr[i+offset] = charArr[i] + base[index];
    }
}